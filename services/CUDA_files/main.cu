#include <hip/hip_runtime.h>
#include <iostream>
#include "small_label.hpp"
#include "kernels.hpp"




extern "C" {
int calculate_label_likelihood(
    double* d_likelihoods, 
    double* d_helper_value_1, 
    double* d_helper_value_2, 
    double alpha, 
    double beta, 
    Small_Label* labels, 
    int num_labels, 
    int im_x,  // Image width
    int im_y   // Image height
)
{
    int size = im_x * im_y;  // Total number of pixels

    // **Allocate GPU memory for boolean predictions array**
    bool* d_predictions;
    hipMalloc(&d_predictions, size * sizeof(bool));

    // **Initialize GPU predictions array to false (0)**
    hipMemset(d_predictions, 0, size * sizeof(bool));

    // **Allocate host-side labels array and copy to GPU**
    Small_Label* d_labels;
    hipMalloc(&d_labels, num_labels * sizeof(Small_Label));
    hipMemcpy(d_labels, labels, num_labels * sizeof(Small_Label), hipMemcpyHostToDevice);

    // **CUDA Kernel: Mark labeled regions in `d_predictions`**
    int blockSize = 256;
    int numBlocks = (num_labels + blockSize - 1) / blockSize;
    mark_predictions<<<numBlocks, blockSize>>>(d_predictions, d_labels, num_labels, im_x, im_y);
    hipDeviceSynchronize(); // Ensure marking is complete

    // **Launch the likelihood update kernel**

    update_likelihoods<<<numBlocks, blockSize>>>(d_likelihoods, d_helper_value_1, d_helper_value_2, d_predictions, alpha, beta, im_x, im_y);
    hipDeviceSynchronize(); // Ensure marking is complete

    // **Free device memory**
    hipFree(d_predictions);
    hipFree(d_labels);

    return 0; // Success
}

int main(){
    return 0;
}

}