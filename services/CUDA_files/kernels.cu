#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "small_label.hpp"

extern "C" {  // Ensure these functions are exposed for C linkage

__global__ void update_likelihoods(
    double* likelihoods, 
    double* helper_value_1, 
    double* helper_value_2, 
    bool* predictions, 
    double alpha, 
    double beta, 
    int im_x, 
    int im_y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < im_x * im_y; i += stride){
        bool prediction = predictions[i];
        helper_value_1[i] *= tgamma(1-prediction+alpha) * tgamma(prediction+beta) / tgamma(1 + alpha + beta);
        helper_value_2[i] *= tgamma(prediction+alpha) * tgamma(1-prediction+beta) / tgamma(1 + alpha + beta);
        likelihoods[i] = helper_value_1[i] / (helper_value_1[i] + helper_value_2[i]);
    }
}

__global__ void mark_predictions(bool* d_predictions, Small_Label* d_labels, int num_labels, int im_x, int im_y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_labels) return;

    Small_Label l = d_labels[idx];

    for (int row = l.top_left_y; row <= l.bot_right_y; row++) {
        for (int col = l.top_left_x; col <= l.bot_right_x; col++) {
            d_predictions[row * im_x + col] = true;
        }
    }
}

// Export function to launch kernels
extern "C" void launch_update_likelihoods(
    double* d_likelihoods, 
    double* d_helper_value_1, 
    double* d_helper_value_2, 
    bool* d_predictions, 
    double alpha, 
    double beta, 
    int im_x, 
    int im_y
) {
    int blockSize = 256;
    int numBlocks = (im_x * im_y + blockSize - 1) / blockSize;
    update_likelihoods<<<numBlocks, blockSize>>>(d_likelihoods, d_helper_value_1, d_helper_value_2, d_predictions, alpha, beta, im_x, im_y);
}

extern "C" void launch_mark_predictions(
    bool* d_predictions, 
    Small_Label* d_labels, 
    int num_labels, 
    int im_x, 
    int im_y
) {
    int blockSize = 256;
    int numBlocks = (num_labels + blockSize - 1) / blockSize;
    mark_predictions<<<numBlocks, blockSize>>>(d_predictions, d_labels, num_labels, im_x, im_y);
}
}
